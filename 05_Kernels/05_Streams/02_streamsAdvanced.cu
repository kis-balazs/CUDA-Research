#include <iostream>
#include <hip/hip_runtime.h>

// CHECK_CUDA_ERROR definition for detailed logging

#define LEN 1000000
#define BLOCK_SIZE 256

__global__ void k1(float *data, int n) {
	int thrIdx = blockIdx.x * blockDim.x + threadIdx.x;

	if (thrIdx < n)
		data[thrIdx] *= 2;
}

__global__ void k2(float *data, int n) {
	int thrIdx = blockIdx.x * blockDim.x + threadIdx.x;

	if (thrIdx < n)
		data[thrIdx] += 1;
}

void CUDART_CB myStreamCallback(hipStream_t stream, hipError_t error, void *userData) {
	std::cout << "Stream callback print!" << std::endl;
}


int main() {
	float *hData, *dData;
	hipStream_t stream1, stream2;
	hipEvent_t event;
	std::cout << event << std::endl;

	size_t size = LEN * sizeof(float);

	hipHostMalloc(&hData, size, hipHostMallocDefault);
	hipMalloc(&dData, size);

	for (int i = 0; i < LEN; i++)
		hData[i] = i;

	int leastPrio, greatestPrio;
	hipDeviceGetStreamPriorityRange(&leastPrio, &greatestPrio);
	hipStreamCreateWithPriority(&stream1, hipStreamNonBlocking, leastPrio);
	hipStreamCreateWithPriority(&stream2, hipStreamNonBlocking, greatestPrio);

	hipEventCreate(&event);

	hipMemcpyAsync(dData, hData, size, hipMemcpyHostToDevice, stream1);

	int gridSize = (LEN + BLOCK_SIZE - 1) / BLOCK_SIZE;
	k1<<<gridSize, BLOCK_SIZE, 0, stream1>>>(dData, LEN);
	
	hipEventRecord(event, stream1);

	// simulate operation priority using events for streams
	hipStreamWaitEvent(stream2, event, 0);

	k2<<<gridSize, BLOCK_SIZE, 0, stream2>>>(dData, LEN);

	hipStreamAddCallback(stream2, myStreamCallback, NULL, 0);

	
	hipMemcpyAsync(hData, dData, size, hipMemcpyDeviceToHost, stream2);

	hipStreamSynchronize(stream1);
	hipStreamSynchronize(stream2);


	bool correct = true;
	for (int i = 0; i < LEN; i++) {
		float expected = i * 2 + 1;
		if (fabs(hData[i] - expected) > 1e-5) {
			correct = false;
			break;
		}
	}
	std::cout << "Results are ";
       	if (correct)
		std::cout << "correct";
	else
		std::cout << "incorrect";
	std::cout << std::endl;

	hipHostFree(hData); hipFree(dData);
	hipStreamDestroy(stream1); hipStreamDestroy(stream2);
	hipEventDestroy(event);

	return 0;
}
