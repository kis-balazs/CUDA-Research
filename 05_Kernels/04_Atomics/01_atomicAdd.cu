#include <stdio.h>
#include <hip/hip_runtime.h>

#define N_THREADS 100
#define N_BLOCKS 100

__global__ void incrCntNonAtomic(int *counter) {
	// not locking mutex
	int old = *counter;
	*counter = old + 1;
	// not unlocking mutex
	
	// normally, old value is returned, e.g., on CAS operations
}

__global__ void incrCntAtomic(int *counter) {
	atomicAdd(counter, 1);
}


int main() {
	int h_cntNonAtomic = 0;
	int h_cntAtomic = 0;
	int *d_cntNonAtomic, *d_cntAtomic;

	size_t size = sizeof(int);

	hipMalloc(&d_cntNonAtomic, size);
	hipMalloc(&d_cntAtomic, size);

	hipMemcpy(d_cntNonAtomic, &h_cntNonAtomic, size, hipMemcpyHostToDevice);
	hipMemcpy(d_cntAtomic, &h_cntAtomic, size, hipMemcpyHostToDevice);

	incrCntNonAtomic<<<N_BLOCKS, N_THREADS>>>(d_cntNonAtomic);
	incrCntAtomic<<<N_BLOCKS, N_THREADS>>>(d_cntAtomic);

	hipMemcpy(&h_cntNonAtomic, d_cntNonAtomic, size, hipMemcpyDeviceToHost);
	hipMemcpy(&h_cntAtomic, d_cntAtomic, size, hipMemcpyDeviceToHost);

	printf("counter [nonAtomic]: \t%d\n", h_cntNonAtomic);
	printf("counter [atomic]: \t%d\n", h_cntAtomic);

	hipFree(d_cntNonAtomic); hipFree(d_cntAtomic);

	return 0;
}
