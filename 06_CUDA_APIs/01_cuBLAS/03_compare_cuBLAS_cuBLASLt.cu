#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipblaslt.h>
#include <hip/hip_fp16.h>
#include <functional>
#include <vector>
#include <numeric>


#define N 512
#define M 256
#define K 512

// CHECK_CUDA_ERROR definition for detailed logging

void initMat(float *MAT, int r, int c) {
    for (int i = 0; i < r * c; i++) MAT[i] = (float)rand() / RAND_MAX;
}

// CUDA event-based timing function
float timeKernel(std::function<void()> kernel) {
	hipEvent_t start, stop;
	float time;

	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start);
	kernel();
	hipEventRecord(stop);
	hipEventSynchronize(stop);

	hipEventElapsedTime(&time, start, stop);

	hipEventDestroy(start);
	hipEventDestroy(stop);

	return time;
}

float benchmarkKernel(std::function<void()> kernel, int runsWarmup, int runsBenchmark) {
	for (int i = 0; i < runsWarmup; i++) kernel();

	std::vector<float> times;
	for (int i = 0; i < runsBenchmark; i++) times.push_back(timeKernel(kernel));

	return std::accumulate(times.begin(), times.end(), 0.0f) / runsBenchmark;
}


__global__ void mulMatsGpu(float *A, float *B, float *C, int n, int m, int k) {
	int row = blockIdx.x * blockDim.x + threadIdx.x;
	int col = blockIdx.y * blockDim.y + threadIdx.y;

	if (row < n && col < k) {
		float sum = 0;
		for (int l = 0; l < m; l++)
			sum += A[row * m + l] * B[l * k + col];
		C[row * k + col] = sum;
	}
}


int main() {
	float A[N * M];
	float B[M * K];

	initMat(A, N, M);
	initMat(B, M, K);
	
	float C[N * K];
	float CcuBLASFp32[N * K], CcuBLASFp16[N * K];
	float CcuBLASLtFp32[N * K], CcuBLASLtFp16[N * K];

	size_t sizeA = N * M * sizeof(float);
	size_t sizeB = M * K * sizeof(float);
	size_t sizeC = N * K * sizeof(float);
	
	size_t sizeAh = N * M * sizeof(half);
	size_t sizeBh = M * K * sizeof(half);
	size_t sizeCh = N * K * sizeof(half);

	float *dA, *dB, *dC;
	hipMalloc(&dA, sizeA);
	hipMalloc(&dB, sizeB);
	hipMalloc(&dC, sizeC);

	half *dAh, *dBh, *dCh;
	hipMalloc(&dAh, sizeAh);
	hipMalloc(&dBh, sizeBh);
	hipMalloc(&dCh, sizeCh);

	hipMemcpy(dA, A, sizeA, hipMemcpyHostToDevice);
	hipMemcpy(dB, B, sizeB, hipMemcpyHostToDevice);

	half Ah[N * M], Bh[M * K];
	for (int i = 0; i < N * M; i++) Ah[i] = __float2half(A[i]);
	for (int i = 0; i < M * K; i++) Bh[i] = __float2half(B[i]);

	hipMemcpy(dAh, Ah, sizeAh, hipMemcpyHostToDevice);
	hipMemcpy(dBh, Bh, sizeBh, hipMemcpyHostToDevice);
	
	hipblasHandle_t handle;
	hipblasCreate(&handle);

	hipblasLtHandle_t handleLt;
	hipblasLtCreate(&handleLt);

	const float alpha = 1.0f, beta = 0.0f; // comes from the generic matmul operation being constructed to mimic linear layer forward
	const float alphah = __float2half(1.0f), betah = __float2half(0.0f);

	const int runsWarmup = 3;
	const int runsBenchmark = 20;

	// -----
       	// naive GPU matrix multiplication
	dim3 blockDim(32, 32);
	dim3 gridDim((K + blockDim.x - 1) / blockDim.x, (N + blockDim.y - 1) / blockDim.y);
	float naiveCUDATime = benchmarkKernel([&]() {
		mulMatsGpu<<<gridDim, blockDim>>>(dA, dB, dC, N, M, K);
	}, runsWarmup, runsBenchmark);
	printf("naive matmul average time: %lf ms\n", naiveCUDATime);
	hipMemcpy(C, dC, sizeC, hipMemcpyDeviceToHost);

	// -----
	// cuBLAS FP32
	float cuBLASFp32Time = benchmarkKernel([&]() {
		hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, K, N, M, &alpha, dB, K, dA, M, &beta, dC, K);
	}, runsWarmup, runsBenchmark);
	printf("cuBLAS FP32 average time: %lf ms\n", cuBLASFp32Time);
	hipMemcpy(CcuBLASFp32, dC, sizeC, hipMemcpyDeviceToHost);
	
	// -----
	// cuBLASLt FP32
	// set up matrix & multiplication descriptors for float32
	hipblasLtMatrixLayout_t lA, lB, lC;
	hipblasLtMatrixLayoutCreate(&lA, HIP_R_32F, M, N, M);
	hipblasLtMatrixLayoutCreate(&lB, HIP_R_32F, K, M, K);
	hipblasLtMatrixLayoutCreate(&lC, HIP_R_32F, M, N, M);

	hipblasLtMatmulDesc_t mmDesc;
	hipblasLtMatmulDescCreate(&mmDesc, HIPBLAS_COMPUTE_32F, HIP_R_32F);

	float cuBLASLtFp32Time = benchmarkKernel([&]() {
		hipblasLtMatmul(handleLt, mmDesc, &alpha, dB, lB, dA, lA, &beta, dC, lC, dC, lC, NULL, NULL, 0, 0);
	}, runsWarmup, runsBenchmark);
	printf("cuBLASLt FP32 average time: %lf ms\n", cuBLASLtFp32Time);
	hipMemcpy(CcuBLASLtFp32, dC, sizeC, hipMemcpyDeviceToHost);

	printf("\n\n");
	bool cuBLASFp32Correct = true, cuBLASLtFp32Correct = true;
	for (int i = 0; i < N * K; i++) {
		if (fabs(C[i] - CcuBLASFp32[i]) > 1e-4) {
			cuBLASFp32Correct = false;
			break;
		}
		if (fabs(C[i] - CcuBLASLtFp32[i]) > 1e-4) {
			cuBLASLtFp32Correct = false;
			break;
		}
	}
	printf("cuBLAS FP32 results are %s\n", cuBLASFp32Correct ? "correct" : "incorrect");
	printf("cuBLAS-Lt FP32 results are %s\n", cuBLASLtFp32Correct ? "correct" : "incorrect");

	return 0;
}
