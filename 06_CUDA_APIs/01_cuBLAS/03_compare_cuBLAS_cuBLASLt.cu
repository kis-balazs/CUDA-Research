#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipblaslt.h>
#include <hip/hip_fp16.h>
#include <functional>
#include <vector>
#include <numeric>


#define N 2048
#define M 1024
#define K 2048

// CHECK_CUDA_ERROR definition for detailed logging

void initMat(float *MAT, int r, int c) {
    for (int i = 0; i < r * c; i++) MAT[i] = (float)rand() / RAND_MAX;
}

// CUDA event-based timing function
float timeKernel(std::function<void()> kernel) {
	hipEvent_t start, stop;
	float time;

	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start);
	kernel();
	hipEventRecord(stop);
	hipEventSynchronize(stop);

	hipEventElapsedTime(&time, start, stop);

	hipEventDestroy(start);
	hipEventDestroy(stop);

	return time;
}

float benchmarkKernel(std::function<void()> kernel, int runsWarmup, int runsBenchmark) {
	for (int i = 0; i < runsWarmup; i++) kernel();

	std::vector<float> times;
	for (int i = 0; i < runsBenchmark; i++) times.push_back(timeKernel(kernel));

	return std::accumulate(times.begin(), times.end(), 0.0f) / runsBenchmark;
}


__global__ void mulMatsGpu(float *A, float *B, float *C, int n, int m, int k) {
	int row = blockIdx.x * blockDim.x + threadIdx.x;
	int col = blockIdx.y * blockDim.y + threadIdx.y;

	if (row < n && col < k) {
		float sum = 0;
		for (int l = 0; l < m; l++)
			sum += A[row * m + l] * B[l * k + col];
		C[row * k + col] = sum;
	}
}


int main() {
	return 0;
}
