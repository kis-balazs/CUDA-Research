#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipblaslt.h>
#include <hip/hip_fp16.h>
#include <functional>
#include <vector>
#include <numeric>


#define N 512
#define M 256
#define K 512

// CHECK_CUDA_ERROR definition for detailed logging

void initMat(float *MAT, int r, int c) {
    for (int i = 0; i < r * c; i++) MAT[i] = (float)rand() / RAND_MAX;
}

// CUDA event-based timing function
float timeKernel(std::function<void()> kernel) {
	hipEvent_t start, stop;
	float time;

	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start);
	kernel();
	hipEventRecord(stop);
	hipEventSynchronize(stop);

	hipEventElapsedTime(&time, start, stop);

	hipEventDestroy(start);
	hipEventDestroy(stop);

	return time;
}

float benchmarkKernel(std::function<void()> kernel, int runsWarmup, int runsBenchmark) {
	for (int i = 0; i < runsWarmup; i++) kernel();

	std::vector<float> times;
	for (int i = 0; i < runsBenchmark; i++) times.push_back(timeKernel(kernel));

	return std::accumulate(times.begin(), times.end(), 0.0f) / runsBenchmark;
}


__global__ void mulMatsGpu(float *A, float *B, float *C, int n, int m, int k) {
	int row = blockIdx.x * blockDim.x + threadIdx.x;
	int col = blockIdx.y * blockDim.y + threadIdx.y;

	if (row < n && col < k) {
		float sum = 0;
		for (int l = 0; l < m; l++)
			sum += A[row * m + l] * B[l * k + col];
		C[row * k + col] = sum;
	}
}


int main() {
	float A[N * M];
	float B[M * K];

	initMat(A, N, M);
	initMat(B, M, K);
	
	float C[N * K];
	float CcuBLASFp32[N * K], CcuBLASFp16[N * K];
	float CcuBLASLtFp32[N * K], CcuBLASLtFp16[N * K];

	size_t sizeA = N * M * sizeof(float);
	size_t sizeB = M * K * sizeof(float);
	size_t sizeC = N * K * sizeof(float);
	
	size_t sizeAh = N * M * sizeof(half);
	size_t sizeBh = M * K * sizeof(half);
	size_t sizeCh = N * K * sizeof(half);

	float *dA, *dB, *dC;
	hipMalloc(&dA, sizeA);
	hipMalloc(&dB, sizeB);
	hipMalloc(&dC, sizeC);

	__half *dAh, *dBh, *dCh;
	hipMalloc(&dAh, sizeAh);
	hipMalloc(&dBh, sizeBh);
	hipMalloc(&dCh, sizeCh);

	hipMemcpy(dA, A, sizeA, hipMemcpyHostToDevice);
	hipMemcpy(dB, B, sizeB, hipMemcpyHostToDevice);

	__half Ah[N * M], Bh[M * K];
	for (int i = 0; i < N * M; i++) Ah[i] = __float2half(A[i]);
	for (int i = 0; i < M * K; i++) Bh[i] = __float2half(B[i]);

	hipMemcpy(dAh, Ah, sizeAh, hipMemcpyHostToDevice);
	hipMemcpy(dBh, Bh, sizeBh, hipMemcpyHostToDevice);
	
	hipblasHandle_t handle;
	hipblasCreate(&handle);

	hipblasLtHandle_t handleLt;
	hipblasLtCreate(&handleLt);

	const float alpha = 1.0f, beta = 0.0f; // comes from the generic matmul operation being constructed to mimic linear layer forward
	const half alphah = __float2half(1.0f), betah = __float2half(0.0f);

	const int runsWarmup = 3;
	const int runsBenchmark = 20;

	// -----
    // naive GPU matrix multiplication
	dim3 blockDim(32, 32);
	dim3 gridDim((K + blockDim.x - 1) / blockDim.x, (N + blockDim.y - 1) / blockDim.y);
	
	float naiveCUDATime = benchmarkKernel([&]() {
		mulMatsGpu<<<gridDim, blockDim>>>(dA, dB, dC, N, M, K);
	}, runsWarmup, runsBenchmark);
	printf("naive matmul average time: %lf ms\n\n", naiveCUDATime);
	hipMemcpy(C, dC, sizeC, hipMemcpyDeviceToHost);

	// -----
	// cuBLAS FP32
	float cuBLASFp32Time = benchmarkKernel([&]() {
		hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, K, N, M, &alpha, dB, K, dA, M, &beta, dC, K);
	}, runsWarmup, runsBenchmark);
	printf("cuBLAS FP32 average time: %lf ms\n", cuBLASFp32Time);
	hipMemcpy(CcuBLASFp32, dC, sizeC, hipMemcpyDeviceToHost);
	
	// -----
	// cuBLASLt FP32
	// set up matrix & multiplication descriptors for float32
	hipblasLtMatrixLayout_t lA, lB, lC;
	hipblasLtMatrixLayoutCreate(&lA, HIP_R_32F, M, N, M);
	hipblasLtMatrixLayoutCreate(&lB, HIP_R_32F, K, M, K);
	hipblasLtMatrixLayoutCreate(&lC, HIP_R_32F, M, N, M);

	hipblasLtMatmulDesc_t mmDesc;
	hipblasLtMatmulDescCreate(&mmDesc, HIPBLAS_COMPUTE_32F, HIP_R_32F);

	// set matrix operation for A and B
	hipblasOperation_t transA = HIPBLAS_OP_N;
	hipblasOperation_t transB = HIPBLAS_OP_N;
	hipblasLtMatmulDescSetAttribute(mmDesc, HIPBLASLT_MATMUL_DESC_TRANSA, &transA, sizeof(hipblasOperation_t));
	hipblasLtMatmulDescSetAttribute(mmDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &transB, sizeof(hipblasOperation_t));

	float cuBLASLtFp32Time = benchmarkKernel([&]() {
		hipblasLtMatmul(handleLt, mmDesc, &alpha, dB, lB, dA, lA, &beta, dC, lC, dC, lC, NULL, NULL, 0, 0);
	}, runsWarmup, runsBenchmark);
	printf("cuBLASLt FP32 average time: %lf ms\n", cuBLASLtFp32Time);
	hipMemcpy(CcuBLASLtFp32, dC, sizeCh, hipMemcpyDeviceToHost);

	bool cuBLASFp32Correct = true, cuBLASLtFp32Correct = true;
	for (int i = 0; i < N * K; i++) {
		if (fabs(C[i] - CcuBLASFp32[i]) > 1e-5) {
			cuBLASFp32Correct = false;
			break;
		}
		if (fabs(C[i] - CcuBLASLtFp32[i]) > 1e-5) {
			cuBLASLtFp32Correct = false;
			break;
		}
	}
	printf("cuBLAS FP32 results are %s\n", cuBLASFp32Correct ? "correct" : "incorrect");
	printf("cuBLAS-Lt FP32 results are %s\n\n", cuBLASLtFp32Correct ? "correct" : "incorrect");

	// -----
	// cuBLAS FP16
	float cuBLASFp16Time = benchmarkKernel([&]() {
		hipblasHgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, K, N, M, &alphah, dBh, K, dAh, M, &betah, dCh, K);
	}, runsWarmup, runsBenchmark);
	printf("cuBLAS FP16 average time: %lf ms\n", cuBLASFp16Time);
	__half Ch[N * K];
	hipMemcpy(Ch, dCh, sizeCh, hipMemcpyDeviceToHost);
	for (int i = 0; i < N * K; i++) CcuBLASFp16[i] = __half2float(Ch[i]);

	// -----
	// cuBLASLt FP16
	// set up matrix & multiplication descriptors for float16
	hipblasLtMatrixLayout_t lAh, lBh, lCh;
    hipblasLtMatrixLayoutCreate(&lAh, HIP_R_16F, M, N, M); // original NMM
    hipblasLtMatrixLayoutCreate(&lBh, HIP_R_16F, K, M, K); // see above
    hipblasLtMatrixLayoutCreate(&lCh, HIP_R_16F, M, N, M);

	hipblasLtMatmulDesc_t mmDesch;
	hipblasLtMatmulDescCreate(&mmDesch, HIPBLAS_COMPUTE_16F, HIP_R_16F);

	// matrix operation for A and B are already set up and used in fp32, just re-use
	hipblasLtMatmulDescSetAttribute(mmDesch, HIPBLASLT_MATMUL_DESC_TRANSA, &transA, sizeof(hipblasOperation_t));
	hipblasLtMatmulDescSetAttribute(mmDesch, HIPBLASLT_MATMUL_DESC_TRANSB, &transB, sizeof(hipblasOperation_t));

	float cuBLASLtFp16Time = benchmarkKernel([&]() {
		hipblasLtMatmul(handleLt, mmDesch, &alphah, dBh, lBh, dAh, lAh, &betah, dCh, lCh, dCh, lCh, NULL, NULL, 0, 0);
	}, runsWarmup, runsBenchmark);
	printf("cuBLASLt FP16 average time: %lf ms\n", cuBLASLtFp16Time);
	
	hipMemcpy(Ch, dCh, sizeC, hipMemcpyDeviceToHost);
	for (int i = 0; i < N * K; i++) CcuBLASLtFp16[i] = __half2float(Ch[i]);

	bool cuBLASFp16Correct = true, cuBLASLtFp16Correct = true;
	for (int i = 0; i < N * K; i++) {
		if (fabs(C[i] - CcuBLASFp16[i]) > 1e-3) {
			cuBLASFp16Correct = false;
			break;
		}
		if (fabs(C[i] - CcuBLASLtFp16[i]) > 1e-3) {
			cuBLASLtFp16Correct = false;
			break;
		}
	}
	printf("cuBLAS FP16 results are %s\n", cuBLASFp16Correct ? "correct" : "incorrect");
	printf("cuBLAS-Lt FP16 results are %s\n\n", cuBLASLtFp16Correct ? "correct" : "incorrect");

	hipblasDestroy(handle);

	hipblasLtDestroy(handleLt);
	hipblasLtMatmulDescDestroy(mmDesc);
	hipblasLtMatrixLayoutDestroy(lA);
	hipblasLtMatrixLayoutDestroy(lB);
	hipblasLtMatrixLayoutDestroy(lC);
	hipblasLtMatmulDescDestroy(mmDesch);
	hipblasLtMatrixLayoutDestroy(lAh);
	hipblasLtMatrixLayoutDestroy(lBh);
	hipblasLtMatrixLayoutDestroy(lCh);

	hipFree(dA); hipFree(dB); hipFree(dC);
	hipFree(dAh); hipFree(dBh); hipFree(dCh);
	return 0;
}
