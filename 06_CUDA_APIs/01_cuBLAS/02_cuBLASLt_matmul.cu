#include <hip/hip_runtime.h>
#include <hipblaslt.h>
#include <hip/hip_fp16.h>

#define N 4
#define M 4
#define K 4

// CHECK_CUDA_ERROR definition for detailed logging

void mulMatsCpu(float *A, float *B, float *C, int n, int m, int k) {
	float sum;
	for (int i = 0; i < n; i++) {
		for (int j = 0; j < k; j++) {
			sum = 0;
			for (int l = 0; l < m; l++)
				sum += A[i * m + l] * B[l * k + j];
			C[i * k + j] = sum;
		}
	}
}

int main() {
	float A[N * M] = {1.0f};
	float B[M * K] = {2.0f};

	float CCpu[N * K], C_fp32[N * K], C_fp16[N * K];

	size_t sizeA = N * M * sizeof(float);
	size_t sizeB = M * K * sizeof(float);
	size_t sizeC = N * K * sizeof(float);
	
	size_t sizeAh = N * M * sizeof(half);
	size_t sizeBh = M * K * sizeof(half);
	size_t sizeCh = N * K * sizeof(half);

	mulMatsCpu(A, B, CCpu, N, M, K);

	float *dA, *dB, *dC;
	hipMalloc(&dA, sizeA);
	hipMalloc(&dB, sizeB);
	hipMalloc(&dC, sizeC);

	half *dAh, *dBh, *dCh;
	hipMalloc(&dAh, sizeAh);
	hipMalloc(&dBh, sizeBh);
	hipMalloc(&dCh, sizeCh);

	hipMemcpy(dA, A, sizeA, hipMemcpyHostToDevice);
	hipMemcpy(dB, B, sizeB, hipMemcpyHostToDevice);

	half Ah[N * M], Bh[M * K];
	for (int i = 0; i < N * M; i++) Ah[i] = __float2half(A[i]);
	for (int i = 0; i < M * K; i++) Bh[i] = __float2half(B[i]);

	hipMemcpy(dAh, Ah, sizeAh, hipMemcpyHostToDevice);
	hipMemcpy(dBh, Bh, sizeBh, hipMemcpyHostToDevice);

	hipblasLtHandle_t handle;
	hipblasLtCreate(&handle);

	// set up matrix & multiplication descriptors for float32
	hipblasLtMatrixLayout_t lA, lB, lC;
	hipblasLtMatrixLayoutCreate(&lA, HIP_R_32F, M, N, M);
	hipblasLtMatrixLayoutCreate(&lB, HIP_R_32F, K, M, K);
	hipblasLtMatrixLayoutCreate(&lC, HIP_R_32F, M, N, M);

	hipblasLtMatmulDesc_t mmDesc;
	hipblasLtMatmulDescCreate(&mmDesc, HIPBLAS_COMPUTE_32F, HIP_R_32F);

	// set up matrix & multiplication descriptors for float16
	hipblasLtMatrixLayout_t lAh, lBh, lCh;
    hipblasLtMatrixLayoutCreate(&lAh, HIP_R_16F, M, N, M); // original NMM
    hipblasLtMatrixLayoutCreate(&lBh, HIP_R_16F, K, M, K); // see above
    hipblasLtMatrixLayoutCreate(&lCh, HIP_R_16F, M, N, M);

	hipblasLtMatmulDesc_t mmDesch;
	hipblasLtMatmulDescCreate(&mmDesch, HIPBLAS_COMPUTE_16F, HIP_R_16F);

	// set matrix operation for A and B
	hipblasOperation_t transA = HIPBLAS_OP_N;
	hipblasOperation_t transB = HIPBLAS_OP_N;
	hipblasLtMatmulDescSetAttribute(mmDesc, HIPBLASLT_MATMUL_DESC_TRANSA, &transA, sizeof(hipblasOperation_t));
	hipblasLtMatmulDescSetAttribute(mmDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &transB, sizeof(hipblasOperation_t));
	hipblasLtMatmulDescSetAttribute(mmDesch, HIPBLASLT_MATMUL_DESC_TRANSA, &transA, sizeof(hipblasOperation_t));
	hipblasLtMatmulDescSetAttribute(mmDesch, HIPBLASLT_MATMUL_DESC_TRANSB, &transB, sizeof(hipblasOperation_t));

	const float alpha = 1.0, beta = 0.0f;  // comes from the generic matmul operation being constructed to mimic linear layer forward
	hipblasLtMatmul(handle, mmDesc, &alpha, dB, lB, dA, lA, &beta, dC, lC, dC, lC, NULL, NULL, 0, 0);

	const half alphah = __float2half(1.0f);
	const half betah = __float2half(0.0f);
	hipblasLtMatmul(handle, mmDesch, &alphah, dBh, lBh, dAh, lAh, &betah, dCh, lCh, dCh, lCh, NULL, NULL, 0, 0);

	
	hipMemcpy(C_fp32, dC, sizeC, hipMemcpyDeviceToHost);

	half Ch[N * K];
	hipMemcpy(Ch, dCh, sizeCh, hipMemcpyDeviceToHost);

	for (int i = 0; i < N * K; i++) C_fp16[i] = __half2float(Ch[i]);

	bool cublasLtFp32Correct = true, cublasLtFp16Correct = true;
	for (int i = 0; i < N * K; i++) {
		if (fabs(CCpu[i] - C_fp32[i]) > 1e-4) {
			cublasLtFp32Correct = false;
			break;
		}
	}
	printf("cuBLAS-Lt FP32 results are %s\n", cublasLtFp32Correct ? "correct" : "incorrect");

	for (int i = 0; i < N * K; i++) {
		if (fabs(CCpu[i] - C_fp16[i]) > 1e-4) {
			cublasLtFp16Correct = false;
			break;
		}
	}
	printf("cuBLAS-Lt FP16 results are %s\n", cublasLtFp16Correct ? "correct" : "incorrect");

	hipFree(dA); hipFree(dB); hipFree(dC);
	hipFree(dAh); hipFree(dBh); hipFree(dCh);
	hipblasLtMatrixLayoutDestroy(lA);
	hipblasLtMatrixLayoutDestroy(lAh);
	hipblasLtMatrixLayoutDestroy(lB);
	hipblasLtMatrixLayoutDestroy(lBh);
	hipblasLtMatrixLayoutDestroy(lC);
	hipblasLtMatrixLayoutDestroy(lCh);
	hipblasLtMatmulDescDestroy(mmDesc);
	hipblasLtMatmulDescDestroy(mmDesch);
	hipblasLtDestroy(handle);

	return 0;

}	
