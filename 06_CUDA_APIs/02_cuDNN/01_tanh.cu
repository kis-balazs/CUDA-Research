#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipDNN.h>
#include <stdio.h>
#include <functional.h>
#include <numeric.h>

#define N 256
#define C 24
#define H 224
#define W 224

// CHECK_CUDA definition for detailed logging

#define CHECK_CUDNN(call) { \
    hipdnnStatus_t err = call; \
    if (err != HIPDNN_STATUS_SUCCESS) { \
        fprintf(stderr, "cuDNN error in file %s, line %i: %s\n", __FILE__, __LINE__, hipdnnGetErrorString(err));
        exit(EXIT_FAILURE);
    } \
}

// CUDA event-based timing function
float timeKernel(std::function<void()> kernel) {
	hipEvent_t start, stop;
	float time;

	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start);
	kernel();
	hipEventRecord(stop);
	hipEventSynchronize(stop);

	hipEventElapsedTime(&time, start, stop);

	hipEventDestroy(start);
	hipEventDestroy(stop);

	return time;
}

float benchmarkKernel(std::function<void()> kernel, int runsWarmup, int runsBenchmark) {
	for (int i = 0; i < runsWarmup; i++) kernel();

	std::vector<float> times;
	for (int i = 0; i < runsBenchmark; i++) times.push_back(timeKernel(kernel));

	return std::accumulate(times.begin(), times.end(), 0.0f) / runsBenchmark;
}


__global__ void tanhGpuNaive(float *i, float *o, int n) {
    int thrIdx = blockIdx.x * blockDim.x + threadIdx.x;

    if (thrIdx < n)
        o[thrIdx] = tanhf(i[thrIdx]);
}

float tanhCpu(float i) {
    return tanhf(i);
}


int main() {
    const int tensorSize = N * C * H * W;

    float *hI, *hONaive, *hOCuDNN, *hOCpu;
    float *dI, *dONaive, *dOCuDNN;

    size_t sizeT = tensorSize * sizeof(float);

    hI = (float*)malloc(sizeT);
    hONaive = (float*)malloc(sizeT);
    hOCuDNN = (float*)malloc(sizeT);
    hOCpu = (float*)malloc(sizeT);

    for (int i = 0; i < tensorSize; i++) hI[i] = (float)rand() / RAND_MAX * 2.0 - 1.0;  // [-1, 1]

    hipMalloc(&dI, sizeT);
    hipMalloc(&dONaive, sizeT);
    hipMalloc(&dOCuDNN, sizeT);

    hipMemcpy(dI, hI, sizeT, hipMemcpyHostToDevice);

    // --- CPU Results ---
    for (int i = 0; i < tensorSize; i++) hOCpu[i] = tanhCpu(hI[i]);

    const int runsWarmup = 3;
    const int runsBenchmark = 20;

    // --- GPU Naive Results ---
    dim3 blockDim(256);
    dim3 gridDim((tensorSize + blockDim.x - 1) / blockDim.x);

    float naiveCUDATime = benchmarkKernel([&]() {
		tanhGpuNaive<<<gridDim, blockDim>>>(dI, dONaive, tensorSize);
	}, runsWarmup, runsBenchmark);
	printf("naive GPU tanh() average time: %lf ms\n\n", naiveCUDATime);
	hipMemcpy(hONaive, dONaive, sizeT, hipMemcpyDeviceToHost);


    bool gpuNaiveCorrect = true, gpuCuDNNCorrect = true;
	for (int i = 0; i < tensorSize; i++) {
		if (fabs(hOCpu[i] - hONaive[i]) > 1e-5) {
			gpuNaiveCorrect = false;
			break;
		}
		// if (fabs(hOCpu[i] - hOCuDNN[i]) > 1e-5) {
		// 	gpuCuDNNCorrect = false;
		// 	break;
		// }
	}
	printf("naive GPU tanh() results are %s\n", gpuNaiveCorrect ? "correct" : "incorrect");
	// printf("cuDNN tanh() results are %s\n\n", gpuCuDNNCorrect ? "correct" : "incorrect");

    return 0;
}